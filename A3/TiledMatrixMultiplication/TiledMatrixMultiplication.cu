#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<time.h>

// Setting tile size
#define TILE_SIZE 16

// Helper function for calculating upper ceil of division
int upper_ceil(int numerator, int denominator) {
    if(numerator % denominator == 0){
        return numerator/denominator;
    }
    return (numerator/denominator) + 1;
}

// Function to check if space can be allocated or not
#define printError(func){                                               \
	hipError_t E  = func;                                              \
	if(E != hipSuccess){                                              	\
		printf( "\nError at line: %d ", __LINE__);                      \
		printf( "\nError:  %s ", hipGetErrorString(E));                \
	}                                                                   \
}                                                                       


// Kernel for matrix multiplication
__global__ void TiledMatrixMultiplication(int *device_A, int *device_B, int *device_C, int m, int n, int k){

    // Calculating row and col value
    int Row = blockIdx.y*TILE_SIZE + threadIdx.y;
    int Col = blockIdx.x*TILE_SIZE + threadIdx.x;

    // Shared memory declared for every block
    __shared__ int shared_A[TILE_SIZE][TILE_SIZE];
    __shared__ int shared_B[TILE_SIZE][TILE_SIZE];

    // Value to be assigned
    int device_C_value = 0;

    // Iterating over tiles
    for (int i = 0; i < (TILE_SIZE + n - 1)/TILE_SIZE; i++) {

        shared_A[threadIdx.y][threadIdx.x] = 0;
        shared_B[threadIdx.y][threadIdx.x] = 0;

        // Values assigned to shared memory by the threads
        if (i*TILE_SIZE + threadIdx.x < n && Row < m)
            shared_A[threadIdx.y][threadIdx.x] = device_A[Row*n + i*TILE_SIZE + threadIdx.x];    

        if (i*TILE_SIZE + threadIdx.y < n && Col < k)
            shared_B[threadIdx.y][threadIdx.x] = device_B[(i*TILE_SIZE + threadIdx.y)*k + Col];
         
        __syncthreads();

        // device_C_value incremented
        for (int j = 0; j < TILE_SIZE; ++j)
            device_C_value += (shared_A[threadIdx.y][j] * shared_B[j][threadIdx.x]);

        __syncthreads();    
    }

    // Assigning the device_C_value
    if (Row < m && Col < k)
        device_C[((blockIdx.y * blockDim.y + threadIdx.y)*k) + (blockIdx.x * blockDim.x)+ threadIdx.x] = device_C_value;
}

// Function to check if result is correct
int check(int m, int n, int k, int *host_A, int *host_B, int *host_C)
{
	int flag=1, row, col, sum, i;	

    for(row= 0;row<m;row++){
        for(col=0;col<k;col++){
            sum=0;
            for(i=0;i<n;i++){
                sum = sum + host_A[row*n + i] * host_B[col + i*k];
			}

			// Checking if the answer is shared_A expected
            if(host_C[row*k + col] != sum){
				flag=0;
				break;
			}
		}
		if(!flag) break;
	}
	
	// Returning flag
    return flag;
}

int main(){

	// Seeding PRNG
	srand(time(NULL));

	int i;

	// Host Matrices
    int *host_A;
    int *host_B;
    int *host_C;

	// Matrix host_A of size (m,n) and Matrix host_B of size (n,k)
    int m = 512;
    int n = 256;
	int k = 512;
	
	// Device matrices
    int *device_A;
    int *device_B;
    int *device_C;

	// Allocating memory
    host_A = (int *)malloc(m * n * sizeof(int));
    host_B = (int *)malloc(n * k * sizeof(int));
    host_C = (int *)malloc(m * k * sizeof(int));

    for(i=0;i<m*n;i++){
		// Assigning values
        host_A[i] = rand()%100;
    }
	
	for(i=0;i<n*k;i++){
		// Assigning values
        host_B[i] = rand()%100;
	}
	
	// Allocating memory with error checking
    printError(hipMalloc((void **)&device_A,  m * n * sizeof(int)));
    printError(hipMalloc((void **)&device_B,  n * k * sizeof(int)));
    printError(hipMalloc((void **)&device_C,  m * k * sizeof(int)));

	// Copying values
    hipMemcpy(device_A, host_A, m * n *  sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_B, host_B, n * k *  sizeof(int), hipMemcpyHostToDevice);

	// Initializing grid size and block size
    dim3 dimGrid(upper_ceil(k,TILE_SIZE), upper_ceil(m,TILE_SIZE), 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);

	// Matrix multiplication
    TiledMatrixMultiplication<<<dimGrid, dimBlock>>>(device_A, device_B, device_C, m, n, k);

	// Copying results
    hipMemcpy(host_C, device_C, m * k * sizeof(int), hipMemcpyDeviceToHost);

	// Checking results
    if(check(m, n, k, host_A, host_B, host_C))
      printf("Correct\n");

    else
       printf("Incorrect\n");


	// Freeing memory
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    free(host_A);
    free(host_B);
	free(host_C);
	
	return 0;
}