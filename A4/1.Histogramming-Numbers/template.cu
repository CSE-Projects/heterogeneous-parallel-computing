#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "wb.h"

// #define NUM_BINS 4096
#define NUM_BINS 4096
#define COUNT_MAX 127

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

/**
 * Kernel to perform the histogramming of the input data of unsigned integers
 */
__global__
void HistogramInt(unsigned *deviceInput, unsigned *deviceBins, int inputLength) {
  
  // calculate index at which this thread will function
  unsigned index = threadIdx.x + blockDim.x * blockIdx.x;
  
  // create a private array of bins for each thread block
  __shared__ unsigned shared_bins[NUM_BINS];

  // each thread will handle a bin initialization starting at its index in a block
  // and then at offsets of (number of threads in a block)
  unsigned i = threadIdx.x;
  while (i < NUM_BINS) {
    shared_bins[i] = 0;
    i += blockDim.x; 
  }
  // wait for all threads to complete initialization
  __syncthreads();

  // perform histogramming of the input data
  i = index;  
  // stide block length is all the threads generated  
  int stride = blockDim.x * gridDim.x;
  // considering the input data to be divide into divisions of stride length
  // here each thread handles a input data starting from its index overall 
  // and then will skip over stride length and take in the next input data from next division
  while (i < inputLength) {
    atomicAdd(&shared_bins[deviceInput[i]], 1);
    i += stride;
  }
  // wait for all threads to complete
  __syncthreads();

  // each thread will handle a private bin transfer to global memory starting at its index in a block
  // and then at offsets of (number of threads in a block)
  i = threadIdx.x;
  while (i < NUM_BINS) {
    atomicAdd(&deviceBins[i], shared_bins[i]);
    i += blockDim.x; 
  }
} 

__global__ 
void CleanData(unsigned *deviceBins) {
  unsigned index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < NUM_BINS) {
    // check if count is over 127
    if (deviceBins[index] > COUNT_MAX) {
      deviceBins[index] = COUNT_MAX;
    }
  }
}

int main(int argc, char *argv[]) {

  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  /* Read input arguments here */
  wbArg_t args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
                                       &inputLength);
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  printf("First 10 input values\n");
  for (int i = 0; i < 10; i++) {
    // hostInput[i] = hostInput[i] % 10;
    printf("%u ", hostInput[i]);
  }
  printf("\n");

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput, inputLength * sizeof(unsigned int));
  hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, inputLength * sizeof (unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(deviceBins, deviceBins, NUM_BINS * sizeof (unsigned int), hipMemcpyHostToDevice);
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // Launch kernel
  // ----------------------------------------------------------
  wbLog(TRACE, "Launching kernel");
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Perform kernel computation here
  HistogramInt<<<10, 64>>> (deviceInput, deviceBins, inputLength);
  CleanData<<<(NUM_BINS - 1)/64 + 1, 64>>> (deviceBins);
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceBins);
  hipFree(deviceInput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  // Verify correctness
  // -----------------------------------------------------
  // wbSolution(args, hostBins, NUM_BINS);
  int num;
  unsigned *eOutput = (unsigned int *)wbImport(wbArg_getInputFile(args, 1),
                                       &num);
  bool diff = false;
  for (int i = 0; i < NUM_BINS; i++) {
      if (eOutput[i] != hostBins[i]) {
        printf("%d: %u %u\n", i, hostBins[i], eOutput[i]);
        diff = true;
        break;
      }
  }
  if (!diff) {
    printf("Solution is correct");
  }
  else {
    printf("Solution doesn't match");
  }

  free(hostBins);
  free(hostInput);
  return 0;
}
